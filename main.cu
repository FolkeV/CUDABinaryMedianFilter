/*
 * main.cu
 *
 *  Created on: May 8, 2019
 *      Author: Folke Vesterlund
 */

#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>

#include "medianFilter.cuh"
#include "utils.hpp"

int main(int argc,char **argv){
	std::string fileName;
	size_t numPixels, numRows, numCols;

	if (argc < 2){
		std::cout << "Usage: "<< argv[0] << " <image file>" << std::endl;
		return(-1);
	}
	fileName = argv[1];

	// Read image
	cv::Mat image;
	image = cv::imread(fileName, CV_LOAD_IMAGE_GRAYSCALE);
	if(!image.data){
		std::cerr << "Couldn't open file" << std::endl;
		return(-1);
	}

	if(!image.isContinuous()){
		std::cerr << "Image is not allocated with continuous data. Exiting..." << std::endl;
		return(-1);
	}
	numCols = image.cols;
	numRows = image.rows;
	numPixels = numRows*numCols;

	// Allocate GPU data
	// Image needs to be padded to remove illegal memory accesses.
	size_t nRows, nCols;
	// Calculate padding size
	util::getPadding(numCols, numRows, &nCols, &nRows);

	size_t charPitch;
	size_t intPitch;
	unsigned char* d_binaryImg;
	unsigned char* d_filteredImg;
	unsigned  int* d_temp;
	unsigned  int* d_temp2;
	hipMallocPitch(&d_binaryImg  , &charPitch, nCols * sizeof(char), nRows);
	hipMallocPitch(&d_temp       , &intPitch , nCols * sizeof(int) , nRows);
	hipMallocPitch(&d_temp2      , &intPitch , nCols * sizeof(int) , nRows);
	// Final image can be stored in managed memory for easier post processing
	hipMallocManaged(&d_filteredImg, numPixels * sizeof(char));

	// Pre process image
	int imgMean = util::mean(image.data, numPixels);
	util::threshold(image.data, image.data, imgMean, numPixels);

	// Copy image to GPU
	hipMemcpy2D(d_binaryImg, charPitch, image.data , numCols*sizeof(char), numCols * sizeof(char), numRows, hipMemcpyHostToDevice);

	// Run kernel
	medianFilter(d_filteredImg, d_binaryImg, d_temp, d_temp2, numCols, numRows,charPitch, intPitch);
	hipDeviceSynchronize();

	// No need to copy image back from GPU as it is handled by managed memory

	// Plot result
	cv::Mat finalImage(numRows, numCols, CV_8UC1, (void*)d_filteredImg);
	cv::imshow("Labelled image", finalImage);
	cv::waitKey();

	// Free memory
	hipFree(d_binaryImg);
	hipFree(d_filteredImg);
	hipFree(d_temp);
	hipFree(d_temp2);
}
